
#include <hip/hip_runtime.h>
#define initialStepBoundFactor 0.05f
#define orthoTolerance 10e-5f
#define costRelativeTolerance 10e-5f
#define parRelativeTolerance 10e-5f
#define X0 0
#define Y0 1
#define SX 2
#define SY 3
#define I0 4
#define BG 5
#define SQRTPI  0.564189584f
#define HIP_SQRT_TWO_F       1.414213562f
#define FLT_EPSILON 1.1920928e-7f
#define FLT_MIN 	1.175494e-38F
#define PARAM_LENGTH 6
#define IMSZBIG 21	
#define qrRankingThreshold FLT_MIN

__device__ float toData(const float* m, const int r, const int c) {
	return m[r*PARAM_LENGTH + c];
}

__device__ void fromData(float* m, const int r, const int c, const float value) {
	//if (m == NULL) return;
	m[r*PARAM_LENGTH + c] = value;
}

__device__ void subtract(const float* v, const float* w, const int dim, float* result) {
	if (v == NULL || w == NULL) return;
	for (int i = 0; i < dim; i++)
		result[i] = v[i] - w[i];
}

__device__ double getCost(float* v, const int dim) {
	double dot = 0.;

	for (int i = 0; i < dim; i++)
		dot += v[i] * v[i];

	return sqrt(dot);
}

// Math functions

__device__ float dErf(const float v) {
	return 2.f * expf(-v*v) * SQRTPI;
}

__device__ float Ex(const uint x, const float tsx, float* variables) {
	float e1 = erff(tsx*(x - variables[X0] + 0.5f));
	float e2 = erff(tsx*(x - variables[X0] - 0.5f));
	return 0.5f*e1 - 0.5f*e2;
}

__device__ float Ey(const uint y, const float tsy, float* variables) {
	float e1 = erff(tsy*(y - variables[Y0] + 0.5f));
	float e2 = erff(tsy*(y - variables[Y0] - 0.5f));
	return 0.5f*e1 - 0.5f*e2;
}

__device__ float dEx(const uint x, const float tsx, float* variables) {
	return 0.5f*tsx*(dErf(tsx*(x - variables[X0] - 0.5f)) - dErf(tsx*(x - variables[X0] + 0.5f)));
}

__device__ float dEy(const uint y, const float tsy, float* variables) {
	return 0.5f*tsy*(dErf(tsy*(y - variables[Y0] - 0.5f)) - dErf(tsy*(y - variables[Y0] + 0.5f)));
}

__device__ float dEsx(const uint x, const float tsx, float* variables) {
	return 0.5f*tsx*((x - variables[X0] - 0.5f)*dErf(tsx*(x - variables[X0] - 0.5f)) - (x - variables[X0] + 0.5f)*dErf(tsx*(x - variables[X0] + 0.5f))) / variables[SX];
}

__device__ float dEsy(const uint y, const float tsy, float* variables) {
	return 0.5f*tsy*((y - variables[Y0] - 0.5f)*dErf(tsy*(y - variables[Y0] - 0.5f)) - (y - variables[Y0] + 0.5f)*dErf(tsy*(y - variables[Y0] + 0.5f))) / variables[SY];
}

__device__ float getValue(float* params, const uint x, const uint y) {
	float tsx = 1.f / (HIP_SQRT_TWO_F*params[SX]);
	float tsy = 1.f / (HIP_SQRT_TWO_F*params[SY]);
	float ex = Ex(x, tsx, params);
	float ey = Ey(y, tsy, params);
	return  params[I0] * ex * ey + params[BG];
}

__device__ void getValues(float* point, const int size, float* retVal) {
	const int length = size*size;

	for (int i = 0; i < length; i++) 
		retVal[i] = getValue(point, i%size, i/size);
}

__device__ void getJacobian(float* point, const int size, float* jacobian) {
	float ex,ey;
	uint x,y,i;
	const float tsx = 1.f / (HIP_SQRT_TWO_F*point[SX]);
	const float tsy = 1.f / (HIP_SQRT_TWO_F*point[SY]);
	
	for (i = 0; i < size*size; ++i) {
		x = i % size;
		y = i / size;
		ex = Ex(x, tsx, point);
		ey = Ey(y, tsy, point);
		fromData(jacobian, i, X0, point[I0] * ey*dEx(x, tsx, point));
		fromData(jacobian, i, Y0, point[I0] * ex*dEy(y, tsy, point));
		fromData(jacobian, i, SX, point[I0] * ey*dEsx(x, tsx, point));
		fromData(jacobian, i, SY, point[I0] * ex*dEsy(y, tsy, point));
		fromData(jacobian, i, I0, ex*ey);
		fromData(jacobian, i, BG, 1.f);
	}
}

__device__	int converged(float* p, float* c) {
	if (abs(p[X0] - c[X0]) > 0.001)return 1;
	if (abs(p[Y0] - c[Y0]) > 0.001)return 2;
	if (abs(p[SX] - c[SX]) > 0.002)return 3;
	if (abs(p[SY] - c[SY]) > 0.002)return 4;
	if (abs(p[I0] - c[I0]) > 0.01)return 5;
	if (abs(p[BG] - c[BG]) > 0.01)return 6;
	return 0;
}

// fitting functions

__device__ void qTy(float* beta, int* permutation, float* wJ, const int nR, float* y) {
	int i,k, pk;
	float gamma;
	for (k = 0; k < PARAM_LENGTH; ++k) {
		pk = permutation[k];
		gamma = 0;
		for (i = k; i < nR; ++i) 
			gamma += toData(wJ,i,pk) * y[i];
		
		gamma *= beta[pk];
		for (i = k; i < nR; ++i) 
			y[i] -= gamma * toData(wJ,i,pk);
		
	}
}

__device__ void qrDecomposition(const int solvedCols, float* diagR, float* jacNorm, float* beta, int* permutation, int* rank, float* jacobian, const int nR, float* wjacobian) {
	const int nC = PARAM_LENGTH;
	float akk,norm2,ak2,aki,gamma,alpha,betak;
	int i,j,k,dk,pk,nextColumn;
		
	// Code in this class assumes that the weighted Jacobian is -(W^(1/2) J),
	// hence the multiplication by -1.
	for (j = 0; j < nR*PARAM_LENGTH; j++)
			wjacobian[j] = -jacobian[j];
	
	// initializations
	for (k = 0; k < nC; ++k) {
		permutation[k] = k;
		norm2 = 0.0f;
		
		for (i = 0; i < nR; ++i) {
			akk = toData(wjacobian,i,k);
			norm2 += akk * akk;
		}
		jacNorm[k] = sqrtf(norm2);
	}

	// transform the matrix column after column
	for (k = 0; k < nC; ++k) {

		// select the column with the greatest norm on active components
		nextColumn = -1;
		ak2 = -INFINITY;
		for (i = k; i < nC; ++i) {
			norm2 = 0.f;
			for (j = k; j < nR; ++j) {
				aki = toData(wjacobian,j,permutation[i]);
				norm2 += aki * aki;
			}
			if (isinf(norm2) || isnan(norm2)) {
				return; //UNABLE_TO_PERFORM_QR_DECOMPOSITION_ON_JACOBIAN;
			}
			if (norm2 > ak2) {
				nextColumn = i;
				ak2 = norm2;
			}
		}
		if (ak2 <= qrRankingThreshold) {
			*rank=k; 
		}
		pk = permutation[nextColumn];
		permutation[nextColumn] = permutation[k];
		permutation[k] = pk;

		// choose alpha such that Hk.u = alpha ek
		akk = toData(wjacobian,k,pk);
		alpha = (akk > 0.f) ? -sqrtf(ak2) : sqrtf(ak2);
		betak = 1.0f / (ak2 - akk * alpha);
		beta[pk] = betak;

		// transform the current column
		diagR[pk] = alpha;
		wjacobian[k*nC+pk] -= alpha;

		// transform the remaining columns
		for (dk = nC - 1 - k; dk > 0; --dk) {
			gamma = 0.0f;
			for (j = k; j < nR; ++j) {
				gamma += toData(wjacobian,j,pk) * toData(wjacobian,j,permutation[k + dk]);
			}
			gamma *= betak;
			for (j = k; j < nR; ++j) {
				wjacobian[j*nC+permutation[k + dk]] -= gamma * toData(wjacobian,j,pk);
			}
		}
	}
	*rank = solvedCols;
	return;
}

__device__ void determineLMDirection(const int solvedCols, float* diagR, int* permutation, float* lmDir, float* weightedJacobian, 
	float* qy, float* diag, float* lmDiag, float* work) {

	int i, j, pj, k, pk;
	float dpj, qtbpj, sin, cos, rkk, tan, cotan, temp, rik, temp2, sum;

	// copy R and Qty to preserve input and initialize s
	//  in particular, save the diagonal elements of R in lmDir
	for (j = 0; j < solvedCols; ++j) {
		pj = permutation[j];
		for (i = j + 1; i < solvedCols; ++i) {
			fromData(weightedJacobian,i,pj, toData(weightedJacobian,j,permutation[i]));
		}
		lmDir[j] = diagR[pj];
		work[j] = qy[j];
	}

	// eliminate the diagonal matrix d using a Givens rotation
	for (j = 0; j < solvedCols; ++j) {

		// prepare the row of d to be eliminated, locating the
		// diagonal element using p from the Q.R. factorization
		pj = permutation[j];
		dpj = diag[pj];
		if (dpj != 0.f) {
			for (k = j + 1; k < PARAM_LENGTH; k++)
				lmDiag[k]=0.f;
		}
		lmDiag[j] = dpj;

		//  the transformations to eliminate the row of d
		// modify only a single element of Qty
		// beyond the first n, which is initially zero.
		qtbpj = 0.f;
		for (k = j; k < solvedCols; ++k) {
			pk = permutation[k];

			// determine a Givens rotation which eliminates the
			// appropriate element in the current row of d
			if (lmDiag[k] != 0.f) {

				rkk = toData(weightedJacobian,k,pk);
				if (abs(rkk) < abs(lmDiag[k])) {
					cotan = rkk / lmDiag[k];
					sin = 1.0f / sqrtf(1.0f + cotan * cotan);
					cos = sin * cotan;
				}
				else {
					tan = lmDiag[k] / rkk;
					cos = 1.0f /sqrtf(1.0f + tan * tan);
					sin = cos * tan;
				}

				// compute the modified diagonal element of R and
				// the modified element of (Qty,0)
				fromData(weightedJacobian, k, pk, cos * rkk + sin * lmDiag[k]);
				temp = cos * work[k] + sin * qtbpj;
				qtbpj = -sin * work[k] + cos * qtbpj;
				work[k] = temp;

				// accumulate the tranformation in the row of s
				for (i = k + 1; i < solvedCols; ++i) {
					rik = toData(weightedJacobian, i, pk);
					temp2 = cos * rik + sin * lmDiag[i];
					lmDiag[i] = -sin * rik + cos * lmDiag[i];
					fromData(weightedJacobian, i, pk, temp2);
				}
			}
		}

		// store the diagonal element of s and restore
		// the corresponding diagonal element of R
		lmDiag[j] = toData(weightedJacobian, j, permutation[j]);
		fromData(weightedJacobian, j, permutation[j], lmDir[j]);
	}

	// solve the triangular system for z, if the system is
	// singular, then obtain a least squares solution
	int nSing = solvedCols;
	for (j = 0; j < solvedCols; ++j) {
		if ((lmDiag[j] == 0.f) && (nSing == solvedCols)) {
			nSing = j;
		}
		if (nSing < solvedCols) {
			work[j] = 0.f;
		}
	}
	if (nSing > 0) {
		for (j = nSing - 1; j >= 0; --j) {
			pj = permutation[j];
			sum = 0.f;
			for (i = j + 1; i < nSing; ++i) {
				sum += toData(weightedJacobian, i, pj) * work[i];
			}
			work[j] = (work[j] - sum) / lmDiag[j];
		}
	}

	// permute the components of z back to components of lmDir
	for (j = 0; j < PARAM_LENGTH; ++j) {
		lmDir[permutation[j]] = work[j];
	}
}

__device__ void determineLMParameter(const int solvedCols, float* diagR, int* permutation, int rank, float* lmPar, float* lmDir, float* weightedJacobian,
	float* qy, const float delta, float* diag, float* work1, float* work2, float* work3) {
	int nC = PARAM_LENGTH;
	float dxNorm, s, fp, ypk;
	int i, j, k, pj, pk;

	// compute and store in x the gauss-newton direction, if the
	// jacobian is rank-deficient, obtain a least squares solution
	for (j = 0; j < rank; ++j) {
		lmDir[permutation[j]] = qy[j];
	}
	for (j = rank; j < nC; ++j) {
		lmDir[permutation[j]] = 0.f;
	}
	for (k = rank - 1; k >= 0; --k) {
		pk = permutation[k];
		ypk = lmDir[pk] / diagR[pk];
		for (i = 0; i < k; ++i) {
			lmDir[permutation[i]] -= ypk * toData(weightedJacobian,i,pk);
		}
		lmDir[pk] = ypk;
	}

	// evaluate the function at the origin, and test
	// for acceptance of the Gauss-Newton direction
	dxNorm = 0.0f;
	for (j = 0; j < solvedCols; ++j) {
		pj = permutation[j];
		s = diag[pj] * lmDir[pj];
		work1[pj] = s;
		dxNorm += s * s;
	}
	dxNorm = sqrtf(dxNorm);
	fp = dxNorm - delta;
	if (fp <= 0.1f * delta) {
		*lmPar = 0.f;
		return;
	}

	// if the jacobian is not rank deficient, the Newton step provides
	// a lower bound, parl, for the zero of the function,
	// otherwise set this bound to zero
	float sum2;
	float sum;
	float parl = 0.f;
	if (rank == solvedCols) {
		for (j = 0; j < solvedCols; ++j) {
			pj = permutation[j];
			work1[pj] *= diag[pj] / dxNorm;
		}
		sum2 = 0.f;
		for (j = 0; j < solvedCols; ++j) {
			pj = permutation[j];
			sum = 0.f;
			for (i = 0; i < j; ++i) {
				sum += toData(weightedJacobian,i,pj) * work1[permutation[i]];
			}
			s = (work1[pj] - sum) / diagR[pj];
			work1[pj] = s;
			sum2 += s * s;
		}
		parl = fp / (delta * sum2);
	}

	// calculate an upper bound, paru, for the zero of the function
	sum2 = 0.f;
	for (j = 0; j < solvedCols; ++j) {
		pj = permutation[j];
		sum = 0.0f;
		for (i = 0; i <= j; ++i) {
			sum += toData(weightedJacobian, i , pj) * qy[i];
		}
		sum /= diag[pj];
		sum2 += sum * sum;
	}
	float gNorm = sqrtf(sum2);
	float paru = gNorm / delta;
	if (paru == 0.f) {
		paru = FLT_MIN / min(delta, 0.1f);
	}

	// if the input par lies outside of the interval (parl,paru),
	// set par to the closer endpoint
	*lmPar = min(paru, max(*lmPar, parl));
	if (*lmPar == 0.f) {
		*lmPar = gNorm / dxNorm;
	}

	int countdown;
	float sPar, previousFP, tmp, correction;
	for (countdown = 10; countdown >= 0; --countdown) {

		// evaluate the function at the current value of lmPar
		if (*lmPar == 0.f) {
			*lmPar = max(FLT_MIN, 0.001f * paru);
		}
		sPar = sqrtf(*lmPar);
		for (j = 0; j < solvedCols; ++j) {
			pj = permutation[j];
			work1[pj] = sPar * diag[pj];
		}
		determineLMDirection(solvedCols,diagR,permutation,lmDir,weightedJacobian,
			qy, work1, work2, work3);

		dxNorm = 0.f;
		for (j = 0; j < solvedCols; ++j) {
			pj = permutation[j];
			s = diag[pj] * lmDir[pj];
			work3[pj] = s;
			dxNorm += s * s;
		}
		dxNorm = sqrtf(dxNorm);
		previousFP = fp;
		fp = dxNorm - delta;

		// if the function is small enough, accept the current value
		// of lmPar, also test for the exceptional cases where parl is zero
		if ((abs(fp) <= 0.1f * delta) ||
			((parl == 0.f) && (fp <= previousFP) && (previousFP < 0.f))) {
			return;
		}

		// compute the Newton correction
		for (j = 0; j < solvedCols; ++j) {
			pj = permutation[j];
			work1[pj] = work3[pj] * diag[pj] / dxNorm;
		}
		for (j = 0; j < solvedCols; ++j) {
			pj = permutation[j];
			work1[pj] /= work2[j];
			tmp = work1[pj];
			for (i = j + 1; i < solvedCols; ++i) {
				work1[permutation[i]] -= toData(weightedJacobian, i, pj) * tmp;
			}
		}
		sum2 = 0.f;
		for (j = 0; j < solvedCols; ++j) {
			s = work1[permutation[j]];
			sum2 += s * s;
		}
		correction = fp / (delta * sum2);

		// depending on the sign of the function, update parl or paru.
		if (fp > 0.f) {
			parl = max(parl, *lmPar);
		}
		else if (fp < 0.f) {
			paru = min(paru, *lmPar);
		}

		// compute an improved estimate for lmPar
		*lmPar = max(parl, *lmPar + correction);
	}
}

//***************************************************************************************************************************
__device__ void kernel_CentroidFitter(const int sz, float *data, float *sx, float *sy,
	float *sx_std, float *sy_std){

	float tmpsx = 0.0f; float tmpsx_std = 0.0f;
	float tmpsy = 0.0f; float tmpsy_std = 0.0f;
	float tmpsum = 0.0f;
	int ii, jj;
	int index = 0;
	float s = 0.f;

	for (jj = 0; jj<sz; jj++)
		for (ii = 0; ii<sz; ii++){
			index = sz*jj + ii;
			s = data[index];
			tmpsx += s * ii;
			tmpsy += s * jj;
			tmpsum += s;
		}

	*sx = tmpsx / tmpsum;
	*sy = tmpsy / tmpsum;

	for (ii = 0; ii<sz; ii++)
		for (jj = 0; jj<sz; jj++) {
			index = sz*jj + ii;
			s = data[index];
			tmpsx_std += s*(ii - *sx)*(ii - *sx);
			tmpsy_std += s*(jj - *sy)*(jj - *sy);
		}

	*sx_std = sqrtf(tmpsx_std / tmpsum / sz);
	*sy_std = sqrtf(tmpsy_std / tmpsum / sz);
}

//***************************************************************************************************************************
extern "C"
__global__ void kernel_LM(float* d_data, uint sz, uint maxIter, uint Nfits, float *d_Parameters) {
	
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int BlockSize = blockDim.x;

	//Prevent read/write past end of array
	if ((bx*BlockSize + tx) >= Nfits) return;
	if (sz > IMSZBIG) return;

	//load data
	float *s_data = d_data + (sz*sz*bx*BlockSize + sz*sz*tx);

	//initial values
	const int nR = sz*sz; // Number of observed data.
	const int nE = IMSZBIG * IMSZBIG; // maximum number of data
	const int nC = PARAM_LENGTH; // Number of parameters.
	float start[nC]; memset(start, 0, nC * sizeof(float));
	kernel_CentroidFitter(sz, s_data, &start[X0], &start[Y0], &start[SX], &start[SY]);
	start[I0] = 65535;
	float lowerBound[nC]={0.f,0.f,0.f,0.f,0.f,0.f};
	float upperBound[nC]={(float)sz,(float)sz,(float)(nR),(float)(nR),65535.f,65535.f};

	uint iterationCounter = 0;
	uint evaluationCounter = 0;

	// arrays shared with the other private methods
	int solvedCols = min(nR, nC);
	float diagR[nC]; memset(diagR, 0, nC * sizeof(float));
	float jacNorm[nC]; memset(jacNorm, 0, nC * sizeof(float));
	float beta[nC]; memset(beta, 0, nC * sizeof(float));
	int permutation[nC]; memset(permutation, 0, nC * sizeof(int));
	float lmDir[nC]; memset(lmDir, 0, nC * sizeof(float));
	float lmPar = 0.f;
	int rank = 0;

	// local point
	float exeption_code = 0.f;
	int pos = (BlockSize*bx + tx)*8;
	float   delta = 0.f;
	float   xNorm = 0.f;
	float diag[nC]; memset(diag, 0, nC * sizeof(float));
	float oldX[nC]; memset(oldX, 0, nC * sizeof(float));
	float oldRes[nE]; memset(oldRes, 0, nE * sizeof(float));
	float qtf[nE]; memset(qtf, 0, nE * sizeof(float));
	float work1[nC]; memset(work1, 0, nC * sizeof(float));
	float work2[nC]; memset(work2, 0, nC * sizeof(float));
	float work3[nC]; memset(work3, 0, nC * sizeof(float));

	float currentValues[nE]; memset(currentValues, 0, nE * sizeof(float));
	float currentResiduals[nE]; memset(currentResiduals, 0, nE * sizeof(float));
	float jacobian[nE*nC]; memset(jacobian, 0, nE*nC*sizeof(float));
	float currentPoint[nC]; memset(currentPoint, 0, nC * sizeof(float));
	float weightedJacobian[nE*nC];
    float weightedResidual[nE];
	float tmpVec[nE]; memset(tmpVec, 0, nE * sizeof(float));
	float previousValues[nE]; memset(previousValues, 0, nE * sizeof(float));
	float previousPoint[nC]; memset(previousPoint, 0, nC * sizeof(float));

	//temporary variables
	int i, j, k, pk, pj;
	float dk, xk, s, r, sum;
	float maxCosine = 0.f;
	float ratio, lmNorm, previousCost, actRed, dirJ, coeff1, coeff2, pc2, preRed, dirDer, tmp;

	// Evaluate the function at the starting point and calculate its norm.
	//value will be reassigned in the loop
	evaluationCounter++;
	getValues(start, sz, currentValues);
	subtract(s_data, currentValues, nR, currentResiduals);
	getJacobian(start, sz, jacobian);
	float currentCost = getCost(currentResiduals, nR);
	memcpy(currentPoint,start,nC*sizeof(float));
	
	// Outer loop.
	bool firstIteration = true;
	while (true) { 
		
		iterationCounter++;
		memcpy(previousPoint, currentPoint, nC * sizeof(float));
		memcpy(previousValues, currentValues, nC * sizeof(float));
		// QR decomposition of the jacobian matrix
		memset(weightedJacobian, 0, nE*nC*sizeof(float));
		qrDecomposition(solvedCols, diagR, jacNorm, beta, permutation, &rank, jacobian, nR, weightedJacobian);

		//residuals already have weights applied
		//memset(weightedResidual, 0, nE * sizeof(float));
		memcpy(weightedResidual, currentResiduals, nR*sizeof(float));

		for (i = 0; i < nR; i++) 
			qtf[i] = weightedResidual[i];
		
		// compute Qt.res
		qTy(beta, permutation, weightedJacobian, nR, qtf);

		// now we don't need Q anymore,
		// so let jacobian contain the R matrix with its diagonal elements
		for (k = 0; k < solvedCols; ++k) {
			pk = permutation[k];
			fromData(weightedJacobian,k,pk,diagR[pk]);
		}

		if (firstIteration) {
			// scale the point according to the norms of the columns
			// of the initial jacobian
			xNorm = 0.f;
			for (k = 0; k < nC; ++k) {
				dk = jacNorm[k];
				if (dk == 0) {
					dk = 1.0f;
				}
				xk = dk * currentPoint[k];
				xNorm += xk * xk;
				diag[k] = dk;
			}
			xNorm = sqrtf(xNorm);

			// initialize the step bound delta
			delta = (xNorm == 0) ? initialStepBoundFactor : (initialStepBoundFactor * xNorm);
		}

		// check orthogonality between function vector and jacobian columns
		maxCosine = 0.f;
		if (currentCost != 0.f) {
			for (j = 0; j < solvedCols; ++j) {
				pj = permutation[j];
				s = jacNorm[pj];
				if (s != 0.f) {
					sum = 0.f;
					for (i = 0; i <= j; ++i) {
						sum += toData(weightedJacobian,i,pj) * qtf[i];
					}
					maxCosine = max(maxCosine, abs(sum) / (s * currentCost));
				}
			}
		}
		// Convergence has been reached.
		if (maxCosine <= orthoTolerance) goto end; 

		// rescale if necessary
		for (j = 0; j < nC; ++j) 
			diag[j] = max(diag[j], jacNorm[j]);

		// Inner loop.
		for (ratio = 0.f; ratio < 1.0e-4f;) {

			// save the state
			for (j = 0; j < solvedCols; ++j) {
				pj = permutation[j];
				oldX[pj] = currentPoint[pj];
			}
			previousCost = currentCost;
			
			memcpy(tmpVec, weightedResidual, nR * sizeof(float));
			memcpy(weightedResidual, oldRes, nR * sizeof(float));
			memcpy(oldRes, tmpVec, nR * sizeof(float));

			// determine the Levenberg-Marquardt parameter
			determineLMParameter(solvedCols, diagR, permutation, rank, &lmPar, lmDir, weightedJacobian,
				qtf, delta, diag, work1, work2, work3);

			// compute the new point and the norm of the evolution direction
			lmNorm = 0.f;
			for (j = 0; j < solvedCols; ++j) {
				pj = permutation[j];
				lmDir[pj] = -lmDir[pj];
				tmp = oldX[pj] + lmDir[pj];
				// kernel bounds
				if (tmp>upperBound[pj]){
					lmDir[pj] = -lmDir[pj];
					currentPoint[pj] = upperBound[pj];
				}
				else if (tmp<lowerBound[pj]){
					lmDir[pj] = -lmDir[pj];
					currentPoint[pj] = lowerBound[pj];
				}else{
					currentPoint[pj] = tmp;
				}
				s = diag[pj] * lmDir[pj];
				lmNorm += s * s;
			}
			lmNorm = sqrtf(lmNorm);
			// on the first iteration, adjust the initial step bound.
			if (firstIteration) {
				delta = min(delta, lmNorm);
			}

			// Evaluate the function at x + p and calculate its norm.
			memset(currentValues, 0, nR * sizeof(float));
			memset(currentResiduals, 0, nR * sizeof(float));
			memset(jacobian, 0, nR*nC * sizeof(float));
			evaluationCounter++;
			getValues(currentPoint, sz, currentValues);
			subtract(s_data, currentValues, nR, currentResiduals);
			getJacobian(currentPoint, sz, jacobian);
			currentCost = getCost(currentResiduals, nR);

			// compute the scaled actual reduction
			actRed = -1.0f;
			if (0.1f * currentCost < previousCost) {
				if (currentCost == previousCost)
					previousCost += 1.0e-6f;
				r = currentCost / previousCost;
				actRed = 1.0f - r * r;
			}

			// compute the scaled predicted reduction
			// and the scaled directional derivative
			for (j = 0; j < solvedCols; ++j) {
				pj = permutation[j];
				dirJ = lmDir[pj];
				work1[j] = 0.f;
				for (i = 0; i <= j; ++i) {
					work1[i] += toData(weightedJacobian,i,pj) * dirJ;
				}
			}
			coeff1 = 0.f;
			for (j = 0; j < solvedCols; ++j) {
				coeff1 += work1[j] * work1[j];
			}
			pc2 = previousCost * previousCost;
			coeff1 /= pc2;
			coeff2 = lmPar * lmNorm * lmNorm / pc2;
			preRed = coeff1 + 2.f * coeff2;
			dirDer = -(coeff1 + coeff2);

			// ratio of the actual to the predicted reduction
			ratio = (preRed == 0.f) ? 0.f : (actRed / preRed);

			// update the step bound
			if (ratio <= 0.25f) {
				tmp = (actRed < 0.f) ? (0.5f * dirDer / (dirDer + 0.5f * actRed)) : 0.5f;
				if ((0.1f * currentCost >= previousCost) || (tmp < 0.1f)) {
					tmp = 0.1f;
				}
				delta = tmp * min(delta, 10.0f * lmNorm);
				lmPar /= tmp;

			}
			else if ((lmPar == 0.f) || (ratio >= 0.75f)) {
				delta = 2.f * lmNorm;
				lmPar *= 0.5f;
			}

			// test for successful iteration.
 			if (ratio >= 1.0e-4f) {
				// successful iteration, update the norm
				firstIteration = false;
				xNorm = 0.f;
				for (k = 0; k < nC; ++k) {
					xk = diag[k] * currentPoint[k];
					xNorm += xk * xk;
				}
				xNorm = sqrtf(xNorm);

				// tests for convergence.
				if((iterationCounter> maxIter)||
					(converged(previousPoint, currentPoint) == 0))
						goto end;
			}
			else {
				// failed iteration, reset the previous values
				if(iterationCounter> maxIter){
					exeption_code = 4; 
					goto exep;
				}
				currentCost = previousCost;
				for (i = 0; i < solvedCols; ++i) {
					pj = permutation[i];
					currentPoint[pj] = oldX[pj];
				}
				memcpy(tmpVec,weightedResidual, nR * sizeof(float));
				memcpy(weightedResidual, oldRes, nR * sizeof(float));
				memcpy(oldRes, tmpVec, nR * sizeof(float));
				// Reset "current" to previous values.
				memcpy(currentValues,previousValues,nE*sizeof(float));
			}

			// Default convergence criteria.
			if ((abs(actRed) <= costRelativeTolerance &&
				preRed <= costRelativeTolerance &&
				ratio <= 2.0f) ||
				delta <= parRelativeTolerance * xNorm) 
				goto end;

			// tests for termination and stringent tolerances
			if (abs(actRed) <= 2.f*FLT_EPSILON &&
				preRed <= 2.f*FLT_EPSILON &&
				ratio <= 2.0f){
					exeption_code = 1;
					goto exep; 
				}
			//throw new ConvergenceException(LocalizedFormats.TOO_SMALL_COST_RELATIVE_TOLERANCE, costRelativeTolerance);

			else if (delta <= 2.f*FLT_EPSILON * xNorm) {
				exeption_code = 2;
				goto exep;
			}
			//throw new ConvergenceException(LocalizedFormats.TOO_SMALL_PARAMETERS_RELATIVE_TOLERANCE, parRelativeTolerance);

			else if (maxCosine <= 2.f*FLT_EPSILON) {
				exeption_code = 3;
				goto exep;
			}
			//throw new ConvergenceException(LocalizedFormats.TOO_SMALL_ORTHOGONALITY_TOLERANCE, orthoTolerance);
		}
	}
//exception released
exep:
	//write to global arrays
	d_Parameters[pos] = 0;
	d_Parameters[pos + 1] = 0;
	d_Parameters[pos + 2] = 0;
	d_Parameters[pos + 3] = 0;
	d_Parameters[pos + 4] = 0;
	d_Parameters[pos + 5] = exeption_code;
	d_Parameters[pos + 6] = (float)iterationCounter;
	d_Parameters[pos + 7] = (float)(bx*BlockSize + tx);
	return;

end:
	//write to global arrays
	d_Parameters[pos] = currentPoint[X0];
	d_Parameters[pos + 1] = currentPoint[Y0];
	d_Parameters[pos + 2] = currentPoint[SX];
	d_Parameters[pos + 3] = currentPoint[SY];
	d_Parameters[pos + 4] = currentPoint[I0];
	d_Parameters[pos + 5] = currentPoint[BG];
	d_Parameters[pos + 6] = (float)iterationCounter;
	d_Parameters[pos + 7] = (float)(bx*BlockSize + tx);
}
